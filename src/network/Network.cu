#include "hip/hip_runtime.h"
#include "../layer/LayerFactory.h"
#include "../loss/LossFactory.h"
#include "../optimize/OptimizeFactory.h"
#include "Network.h"


Network::Network(std::string loss_type, std::string optimize_type) :
  num_batches(0)
{
  this->layer_factory = LayerFactory::get_instance();
  this->loss_func = LossFactory::get_instance()->create(loss_type);
  this->optimize_func = OptimizeFactory::get_instance()->create(optimize_type);
}

Network::~Network()
{
  for (auto layer : layers)
  {
    delete layer;
  }

  for (auto feature : features)
  {
    delete feature;
  }
}

void Network::add_batch(Neurons features, Neurons classes)
{
  batches.push_back(features);
  actuals.push_back(classes);
  num_batches += 1;
}

void Network::add_layer(std::string layer_type)
{
  this->layers.push_back(this->layer_factory->create(layer_type));
}

void Network::add_layer(std::string layer_type, size_t x, size_t y)
{
  this->layers.push_back(this->layer_factory->create(layer_type,
    Dim(x, y, this->batch_size)));
}

std::string classify(Neurons& prediction)
{
  prediction->memcpy_device_to_host();

  float max = prediction[0];
  for (size_t x = 1; x < prediction.dim.x * prediction.dim.y; x++)
  {
    if (max < prediction[x])
    {
      max = prediction[x];
    }
  }

  return this->classes[max];
}

int Network::get_num_classes()
{
  return this->classes.size();
}

void Network::train(int num_epochs, float learning_rate, int checkpt = 100)
{
  for (int e = 0; e < num_epochs; e++)
  {
    float loss = 0.0;
    for (int b = 0; b < num_batches; b++)
    {
      output = forward_propagate(batches.at(b));
      back_propagate(output, batches.at(b), learning_rate);
      float loss += this->loss_func->calculate(output, actuals.at(b));
    }

    if (e % checkpt == 0)
    {
      printf("Epoch: %d, Cost: %f", e, loss / num_batches);
    }
  }
}

Neurons& forward_propagate(Neurons& feature)
{
  Neurons layer_output = feature;
  for (auto layer : layers)
  {
    layer_output = layer->forward_prop(layer_output);
  }

  return layer_output;
}

void back_propagate(Neurons& prediction, Neurons& actual)
{

  //TO DO: calculate loss
  Neurons err = this->loss_func->calculate_deriv(prediction, actual);

  for (std::vector<int>::reverse_iterator i = layers.rbegin();
    i != layers.rend(); i++)
  {
    err = *i->back_prop(err, learning_rate);
  }
}
