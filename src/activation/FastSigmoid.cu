#include "hip/hip_runtime.h"

#include "ActivationFunction.h"
#include "FastSigmoid.h"

#include <cstdlib>
#include <math.h>

namespace neural_network {

  FastSigmoid::FastSigmoid();

  FastSigmoid::~FastSigmoid();

  /// <summary>Approximated Sigmoid activation function.</summary>
  /// <param name="x">Represents the aggregated value obtained during
  /// forward propagation (bias + sum(inputs * weights)) for a given neuron
  /// </param name>
  /// <returns>Returns an approximation of the sigmoid function.</returns>
  double FastSigmoid::calculate(double& x)
  {
    return x / 1 + fabs(x);
  }

  /// <summary>Derivative of approximated Sigmoid activation function.
  /// </summary>
  /// <param name="x">Represents the aggregated value obtained during
  /// forward propagation (bias + sum(inputs * weights)) for a given neuron.
  /// </param name>
  /// <returns>Returns derivative of approximated sigmoid.</returns>
  double FastSigmoid::calculate_deriv(double& x)
  {
    double approx_sigmoid = this->calculate(x);

    return approx_sigmoid * (1 - approx_sigmoid);
  }

}
